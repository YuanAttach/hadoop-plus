#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>  
#include <math.h>
#include <map>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "hip/hip_runtime.h"
#include "knn.h"
#include <time.h>

using namespace std;

#define TRAIN 1
#define TEST 0

__device__ __constant__ int dimPerThread;
double *trains, *tests;
long *trainKeys, *testKeys;
int dimension = 0;
int dataset = TRAIN;
int test_num = 0;
int train_num = 0;
int _debug = 1;
long trainsmem_size;
hipDevice_t  cuDevice=0;

/***********************  init_env_gpu ********************/
//malloc global space
jint init_env_gpu(char* src,const int order,const char* args)
{
//	long gpu_size = getLong(src,0,order,0);
	trainsmem_size = getLong_Simple(src,0,order);
        cuDevice=getInt(src,8,order,0);
	hipSetDevice(cuDevice);

        int _debug = 1;
	if(_debug){
		printf("init_env_gpu to %d device\n", cuDevice);	
		printf("length ..... %d\n",strlen(args));
		if(args != NULL&& strlen(args)!=0)
			printf("init command args :%s\n",args);
	}
	if(args!=NULL && strlen(args)!=0){
		map<string,string> args_map;
		parse_to_map(args,args_map);
//		dimension = getDimension(args_map["dimension"]);
	}
        if(trainsmem_size < (long)(2048 * 1024 * 1024)	)
	{
	    printf("to malloc %d bytes for trains\n", (int)trainsmem_size);
            trains = (double *)malloc((int)trainsmem_size);
            trainKeys = (long *)malloc((int)trainsmem_size);
	}
	else
	{
	    printf("to malloc %d bytes for trains\n", (int)(trainsmem_size - SIZE_OF_DOUBLE));
            trains = (double *)malloc((int)(trainsmem_size - SIZE_OF_DOUBLE));
            trainKeys = (long *)malloc((int)(trainsmem_size - SIZE_OF_DOUBLE));
	}
	if(trains == NULL)
	    printf("malloc addr for trains failed!\n");
	printf("init_env_gpu. \tsuccess!\n");
	return ERR_CODE_OK;
}

/*************** free_mem_gpu *******************/
jint free_mem_gpu()
{
	return ERR_CODE_OK;
}
/************************ put_data_gpu *******************/

jint put_data_gpu(const char *args,const vector<int> input_offset,char *input_data,const int input_limit,const int order , unsigned int *global_input_mem_size)
{
	int _debug = 0;
	if(_debug){
		printf("put_data_gpu ,\tthe buffer limit is %d.\n",input_limit);
	}
	map<string,string> cmd_args;
	if(args != NULL){
		parse_to_map(args,cmd_args);
		if(cmd_args.size()>0){
                    dimension = getDimension(cmd_args["dimension"]);
		    dataset = getDataset(cmd_args["dataset"]);
		    _debug = getDebug(cmd_args["isDebug"]);
		}
	}
	if(_debug){
		printf("input offset size is %d.\t global mem size is %d\n",input_offset.size(),*global_input_mem_size);
	}
	switch(dataset){
	    case TRAIN:
	    {
//		if(trainKeys==NULL)
//		{
//	            printf("to malloc %d bytes for trainKeys\n", (int)(trainsmem_size/dimension));
//         	    trainKeys = (long *)malloc((int)(trainsmem_size/dimension));
//		}
//	        if(trainKeys == NULL)
//	            printf("malloc addr for trainKeys failed!\n");
		int case_num = input_offset.size() ;
		if(trains == NULL)
		{
		trains = (double *)malloc(case_num * dimension * sizeof(double));
		trainKeys = (long *)malloc(case_num * sizeof(double));
		}
	    	if(_debug)
	    	    cout << "put " << case_num << " training vectors " << endl;
		for(int idx = 1; idx <= case_num; idx++)
		{
                    int vector_start = input_offset[idx - 1];
		    if(_debug)
		    {
			cout << "vector start at : " << vector_start << endl;
		    }
		    trainKeys[train_num] = getLong_Simple(input_data, vector_start, order);
		    memcpy(&trains[train_num * dimension], input_data + vector_start + SIZE_LONG, dimension * SIZE_OF_DOUBLE);
//		    for (int i = 0; i < dimension; i++)
//		        trains[train_num * dimension + i] = getDouble_Simple(input_data, vector_start + SIZE_LONG + i * SIZE_OF_DOUBLE, order);
//	            case_num--;
//		    if(_debug)
//		    {
//			for(int i = 0; i < dimension; i++) 
//			    cout << trains[train_num * dimension + i] << " ";
//			cout << endl;
//		    }
		    train_num++;
		}
	   	*global_input_mem_size += input_limit;
	    	return ERR_CODE_OK;
	    }
	    case TEST:
	    {
		int case_num = input_offset.size();
	    	if(_debug)
		    cout << "put " << case_num << "tests" << endl;
		tests = (double *)malloc(case_num * dimension * sizeof(double));
		testKeys = (long *)malloc(case_num * sizeof(double));
		for(int idx = 1; idx <= case_num; idx++)
		{
                    int vector_start = input_offset[idx - 1];
		    testKeys[test_num] = getLong_Simple(input_data, vector_start, order);
		    for (int i = 0; i < dimension; i++)
		        tests[(test_num ) * dimension + i] = getDouble_Simple(input_data, vector_start + SIZE_LONG + i * SIZE_OF_DOUBLE, order);
//	            case_num--;
//		    if(_debug)
//		    {
//			cout << "test_key: " << testKeys[test_num] << ", vector: ";
//			for(int i = 0; i < dimension; i++)
//			    cout << tests[test_num * dimension + i] << " ";
//			 cout << endl; 
//	 	    }
		    test_num++;
		}
	    	*global_input_mem_size += input_limit;
 	     	return ERR_CODE_OK;
	    }
	    default:
	    {
	    	//nothing
	    	//may do the default trans the data which is without process data
	    	break;
 	     }
 	} 
//	if(_debug)
//		printf("after trans data ,the local_size is %d.\n " , local_size );
	return ERR_CODE_OK;
}

/************************  calc_gpu *********************/

jint calc_gpu(const char *args,const int order,const unsigned int g_size,char *dst,int *dst_size,const int dst_capacity)
{
    cerr << "beginning calc gpu." << endl;
    map<string,string> cmd_args;
    int k = 1;
    if(args != NULL){
    	if(_debug){
    		printf("calc args : %s\n",args);
    	}
    	parse_to_map(args,cmd_args);
    	if(cmd_args.size()>0){
    	    dimension = getDimension(cmd_args["dimension"]);
    	    k = getK(cmd_args["k"]);
    	    _debug = getDebug(cmd_args["isDebug"]);
    	}
    }
    jint rs = ERR_CODE_OK;
    if(_debug)
        cerr << "initial dst_size: " << *dst_size << ", there are total " << test_num << " test cases" << endl;;
    double* dist_host;
    int*   ind_host;                 // Pointer to index array
    dist_host   = (double *) malloc(test_num * k * sizeof(double));
    ind_host    = (int *)   malloc(test_num * k * sizeof(double));
//    cout << "tests:" << endl;
//    for(int i = 0; i < test_num; i++)
//    {
//	for (int j = 0; j < dimension; j++)
//	    cout << tests[i * dimension + j] << " ";
//	cout << endl;
//    }
//    cout << "trains:" << endl;
//    for(int i = 0; i < k; i++)
//    {
//	for (int j = 0; j < dimension; j++)
//	    cout << trains[i * dimension + j] << " ";
//	cout << endl;
//    }
//    for(int i = 0; i < test_num; i++)
//    {
//	for(int j = 0; j < k; j++)
//	{
//	    double sum = 0.0;
//	    double psum = 0.0;
//	    for(int nn = 0; nn < 8; nn++)
//	    {
//		psum = 0.0;
//		for(int c = 0; c < dimPerThread; c++)
//		    psum += tests[i * dimension + nn * dimPerThread + c] * trains[j * dimension + nn * dimPerThread + c];
//		printf("testId: %d, trainId: %d, part: %d, psum: %lf\n", i, j, nn, psum);
//		sum += psum;
//	    }
//	    printf("testId: %d, trainId: %d, sum: %lf\n", i, j, sum);
//	}
//    }
    findingknn(trains, tests, ind_host, dist_host, dimension, k, dst, dst_size);
    free(ind_host);
    free(dist_host);
    return rs;
}
//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//
__host__ __device__ inline static
double shddot(int numCoords,
		double *tests, 
		double *trains,
		int trainId,
		int blockDimx,
		int threadId)
{
	int i;
	double ans = 0.0;
//	for(i = 0; i < dimPerThread; i++) {
	for(i = 0; i < 16; i++) {
//		ans += (tests[i * 32] * trains[trainId * numCoords + (threadId / 32) * dimPerThread + i]);
		ans += (tests[i * 32] * trains[trainId * numCoords + (threadId / 32) * 16 + i]);
//		printf("%d, %lf , %lf\n", i, tests[i * 32], trains[trainId * numCoords + (threadId / 32) * dimPerThread + i]);
	}
	return ans;
}
/*----< find_k_nearest_nei() >---------------------------------------------*/
__global__ static
void findKNN(int numCoords,
                           int test_nb, 
			  int train_nb,
                          int k,
                          double *tests,     
                          double *trains,    
			  double *dist_dev,
                           int *ind_dev)
{

	/*in fact, yIndex is always 0,beacuse our block and grid is one dimemsion*/
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int yIndex = threadIdx.y + blockIdx.y * blockDim.y;

    int tId = xIndex + yIndex * gridDim.x * blockDim.x;
    int testId = tId / blockDim.x * 32 +tId % 32;
    extern __shared__ int s[];
    if (testId < test_nb) {
//    printf("threadIdx.x: %d, threadIdx.y: %d, blockIdx.x: %d, blockIdx.y: %d, xIndex: %d, yIndex: %d, testId: %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, xIndex, yIndex, testId);
        int   index, i, min_ind, j; 
        double dist, min_dist;
	int *inds;
	double *dists;
	double *tempDists;
        double *stest;
//	stest = (double *)(s) + (threadIdx.x/32) * dimPerThread * 32 + threadIdx.x % 32;
	stest = (double *)(s) + (threadIdx.x/32) * 16 * 32 + threadIdx.x % 32;
//        for(int i = 0; i < dimPerThread; i++)
        for(int i = 0; i < 16; i++)
        {
//            stest[i  * 32] = tests[testId * numCoords + i + (threadIdx.x / 32) * dimPerThread];
            stest[i  * 32] = tests[testId * numCoords + i + (threadIdx.x / 32) * 16];
        }
	__syncthreads();
	tempDists = (double *)s + numCoords * 32  + threadIdx.x;
//	    printf("test_nb: %d, train_nb: %d, numCoords: %d, k: %d\n", test_nb, train_nb, numCoords, k);
        /* find the cluster id that has min distance to test */
        for(i = 0; i < k; i++)
	{

	    dist =  shddot(numCoords, stest, trains, i, blockDim.x, threadIdx.x);
//            printf("test_id: %d, train_id: %d, threadId: %d, d: %lf\n", testId, i, threadIdx.x, dist);
	    tempDists[0] = dist;
	    __syncthreads();
	    if(threadIdx.x < 128)
                tempDists[0] += tempDists[128];
	    __syncthreads();
	    if(threadIdx.x < 64)
                tempDists[0] += tempDists[64];
	    __syncthreads();
	    if(threadIdx.x < 32)
	    {
                tempDists[0] += tempDists[32];
//		printf("test_id: %d, train_id: %d, distance: %lf\n", testId, i, tempDists[0]);
	        dists = (double *)s + numCoords * 32 + blockDim.x + threadIdx.x;
	        inds = s + (numCoords * 32 + k * 32 + blockDim.x) * 2 + threadIdx.x;
                dists[i * 32] = tempDists[0];
	        inds[i * 32] = i;
	    }
	}
        for (i=k; i<train_nb; i++) {
            dist = shddot(numCoords, stest, trains, i, blockDim.x, threadIdx.x);
	    tempDists[0] = dist;
	    __syncthreads();
	    if(threadIdx.x < 128)
                tempDists[0] += tempDists[128];
	    __syncthreads();
	    if(threadIdx.x < 64)
                tempDists[0] += tempDists[64];
	    __syncthreads();
	    if(threadIdx.x < 32)
	    {
                tempDists[0] += tempDists[32];
//		printf("test_id: %d, train_id: %d, distance: %lf\n", testId, i, tempDists[0]);
	        dists = (double *)s + numCoords * 32 + blockDim.x + threadIdx.x;
	        inds = s + (numCoords * 32 + k * 32 + blockDim.x) * 2 + threadIdx.x;
	        min_dist = dists[0];
	        min_ind = 0;
	        for(index = 1; index < k; index++) 
	        {
	            if(dists[index * 32] < min_dist)
	            {
	                min_dist = dists[index * 32];
	                 min_ind = index;
	            } 
	        }
                if (tempDists[0] > min_dist) { /* find the min and its array index */
                    dists[min_ind * 32] = tempDists[0];
                    inds[min_ind  * 32]  = i; 
                }
            }
        }
	if(threadIdx.x < 32)
	{
//		printf("dimPerThread: %d\n", dimPerThread);
	        dists = (double *)s + numCoords * 32 + blockDim.x + threadIdx.x;
	        inds = s + (numCoords * 32 + k * 32 + blockDim.x) * 2 + threadIdx.x;
            for(i = 0; i < k; i++)
	    {
                dist_dev[testId + i* test_nb] = dists[i* 32 ];
                ind_dev[testId + i * test_nb] = inds[i * 32 ];
//	    if(testId < 32)
//		printf("testId: %d, trainId: %d, dist: %lf\n", testId, ind_dev[testId + i * test_nb], dist_dev[testId + i * test_nb]);
	    }
	}
    }
}
int findingknn(double *ref, double *query, int *ind_host, double *dist_host, int dimension, int k, char *dst, int *dst_size)
{
	double* ref_dev = 0;
	double* query_dev = 0;
        double* dist_dev = 0;                // Pointer to distance array
	int* ind_dev = 0;
	jint rs = ERR_CODE_OK;
        int       max_nb_query_traited;
        size_t       actual_nb_query_width;
        size_t memory_total;

    struct timeval ustart, uend;
    gettimeofday(&ustart, NULL);

	if(_debug)
	    cerr << "dimension at the begining of findingknn: " << dimension << endl;
        // CUDA Initialisation
//	hipEvent_t debug_start, debug_stop;
//	hipEventCreate(&debug_start);
//	hipEventCreate(&debug_stop);
//	hipEventRecord(debug_start, 0);
//	hipFree(0);
//	hipEventRecord(debug_stop, 0);
//	hipEventSynchronize(debug_stop);
//
//    	float debug_costtime;
//        hipEventElapsedTime(&debug_costtime, debug_start, debug_stop);
//	printf("initial time: %f\n", debug_costtime);

//	hipDeviceReset();
//        hipInit(0);
//	hipDeviceSynchronize();
//    gettimeofday(&uend, NULL);
//    int utime = 1000000 * (uend.tv_sec - ustart.tv_sec) + (uend.tv_usec - ustart.tv_usec);
//    printf("time used for hipInit is: %d us\n", utime);
//    gettimeofday(&ustart, NULL);

        max_nb_query_traited = min( test_num, 8 * 1024 * 64);
	if(_debug)
	    cerr << "totally " << train_num << " train cases, max_nb_query_traited = " << max_nb_query_traited << endl;
        /* Allocate device memory for the matrices */
 
        hipMalloc((void **)&ref_dev,  dimension * train_num * sizeof(double));
        hipMalloc((void **)&query_dev, max_nb_query_traited * dimension * sizeof(double)); 
        hipMalloc((void **)&dist_dev, max_nb_query_traited * k * sizeof(double)); 
        hipMalloc((void **)&ind_dev, max_nb_query_traited * k * sizeof(int)); 
	hipDeviceSynchronize();
    gettimeofday(&uend, NULL);
    int utime = 1000000 * (uend.tv_sec - ustart.tv_sec) + (uend.tv_usec - ustart.tv_usec);
    printf("time used for cuda malloc knn is: %d us\n", utime);
    gettimeofday(&ustart, NULL);
 
	hipMemcpy(ref_dev, ref, train_num * dimension * sizeof(double), hipMemcpyHostToDevice);
        hipMemset(dist_dev, 0, max_nb_query_traited * k * sizeof(double));
//	    cerr << hipGetErrorString(hipGetLastError()) << endl;
        for (int i=0; i<test_num; i+=max_nb_query_traited){
            
            	// Number of query points considered
        actual_nb_query_width = min( max_nb_query_traited, (int)test_num - i );
        
        // Copy of part of query actually being treated
	hipMemcpy( (void *)query_dev, ((char *)query) + i * dimension * SIZE_OF_DOUBLE, actual_nb_query_width * dimension * sizeof(double), hipMemcpyHostToDevice);
//	cerr << hipGetErrorString(hipGetLastError()) << endl;
//	int tpb = (49152 / (12 * k + dimension * 8) ) / 32 * 32;
//	if(tpb > 192)
//	    tpb = 192;
	int tpb = 256;
        dim3 g_512x1((actual_nb_query_width + 32 - 1)/32, 1, 1);
        dim3 t_512x1(tpb, 1, 1);
        cerr << "launching kernel with " << (actual_nb_query_width + 32 - 1)/32 << ", " << tpb << endl; 
	hipDeviceSynchronize();
    gettimeofday(&uend, NULL);
    utime = 1000000 * (uend.tv_sec - ustart.tv_sec) + (uend.tv_usec - ustart.tv_usec);
    printf("time used for preparing calc knn is: %d us\n", utime);
    gettimeofday(&ustart, NULL);
	int dpt = dimension / 8;
	hipMemcpyToSymbol(HIP_SYMBOL(dimPerThread), &dpt, sizeof(int), 0, hipMemcpyHostToDevice);
	findKNN<<<g_512x1, t_512x1, (12 * k + dimension * 8)* 32 + tpb * 8>>>(dimension, actual_nb_query_width, train_num, k, query_dev, ref_dev, dist_dev, ind_dev);
	hipDeviceSynchronize();
    gettimeofday(&uend, NULL);
    utime = 1000000 * (uend.tv_sec - ustart.tv_sec) + (uend.tv_usec - ustart.tv_usec);
    printf("time used for knn kernel is: %d us\n", utime);
    gettimeofday(&ustart, NULL);
	cerr << hipGetErrorString(hipGetLastError()) << endl;
	hipMemcpy((void *)dist_host, (void *)dist_dev, max_nb_query_traited * k * sizeof(double), hipMemcpyDeviceToHost);
	cerr << hipGetErrorString(hipGetLastError()) << endl;
	hipMemcpy((void *)ind_host, (void *)ind_dev, max_nb_query_traited * k * sizeof(int), hipMemcpyDeviceToHost);
	cerr << hipGetErrorString(hipGetLastError()) << endl;
	hipDeviceSynchronize();
//	if(_debug)
//	{
//            for(int  ii = 0; ii < actual_nb_query_width; ii++)
//	    {
//                for(int kk = 0; kk < k; kk++)
//                {
//	             printf("testIdx: %d, trainInx: %d, dist_host[%d]: %lf ", i + ii, ind_host[kk * actual_nb_query_width + ii], kk, dist_host[actual_nb_query_width * kk + ii]);
//	        }
//	          printf("\n");
//	    }  
//	}
        for(int ii = 0; ii < actual_nb_query_width; ii++)
            for(int kk = 0; kk < k; kk++) 
            {
//	        _debug = 1;
//                if(_debug)
//	        {
//                    cerr << "putting testKey: " << testKeys[i + ii];
//                    cerr << ", trainIndex: " << ind_host[kk * actual_nb_query_width + ii];
//                    cerr << ", trainKey: " << trainKeys[ind_host[actual_nb_query_width * kk + ii]];
//                    cerr << ", d: " << dist_host[actual_nb_query_width * kk + ii] << "to outputbuffer" << endl;
//	        }
                memcpy(dst + *dst_size, &testKeys[i + ii], SIZE_OF_LONG);
                *dst_size += SIZE_OF_LONG;
                memcpy(dst + *dst_size, &trainKeys[ind_host[actual_nb_query_width * kk + ii]], SIZE_OF_LONG);
                *dst_size += SIZE_OF_LONG;
                memcpy(dst + *dst_size, &dist_host[actual_nb_query_width * kk + ii], SIZE_OF_DOUBLE);
                *dst_size += SIZE_OF_DOUBLE;
	    } 
	}
    gettimeofday(&uend, NULL);
    utime = 1000000 * (uend.tv_sec - ustart.tv_sec) + (uend.tv_usec - ustart.tv_usec);
    printf("time used for cp result is: %d us\n", utime);
	cerr << "dst_size: " << *dst_size << endl; 
        return rs;  
}
